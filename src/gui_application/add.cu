
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y, float *z)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
  {
    z[i] = x[i] * y[i] * y[i];
  }
}

__global__
void step(int n , float *x, float *v, float *a, float *next_x, float *next_v)
{
  constexpr float ts = 0.1f;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
  {
    next_x[i] = x[i] + ts * v[i] + (ts * ts) / 2.0 * a[i];
    next_x[i] = x[i] + ts * v[i];
    next_v[i] = v[i] + ts * a[i];
  }
}

int main(void)
{
  int N = 1<<20;
  float *x, *y, *z, *next_x, *next_v;

  
  // Allocate Unified Memory – accessible from CPU or GPU
  gpuErrchk(hipMallocManaged(&x, N*sizeof(float)));
  gpuErrchk(hipMallocManaged(&y, N*sizeof(float)));
  gpuErrchk(hipMallocManaged(&z, N*sizeof(float)));
  gpuErrchk(hipMallocManaged(&next_x, N*sizeof(float)));
  gpuErrchk(hipMallocManaged(&next_v, N*sizeof(float)));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
    z[i] = 1.0f;
    next_x[i] = 1.0f;
    next_v[i] = 1.0f;
  }

  auto start = std::chrono::high_resolution_clock::now();

  // Run kernel on 1M elements on the GPU
  int blockSize = 512;
  int numBlocks = (N + blockSize - 1) / blockSize;
  step<<<numBlocks, blockSize>>>(N, x, y, z, next_x, next_v);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  auto finish = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = finish - start;
  std::cout << "Elapsed time: " << elapsed.count() << " s\n";

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
