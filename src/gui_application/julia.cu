#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex
{
  float r;
  float i;
  __device__ hipComplex(float a, float b) : r(a), i(b) {}
  __device__ float magnitude2(void)
  {
    return r * r + i * i;
  }
  __device__ hipComplex operator*(const hipComplex &a)
  {
    return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
  }
  __device__ hipComplex operator+(const hipComplex &a)
  {
    return hipComplex(r + a.r, i + a.i);
  }
};

__device__ int julia(int x, int y)
{
  const float scale = 1.5;
  // Convert int coordinates to float coordinates
  float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
  float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

  // Iterate to compute the julia set
  hipComplex c(-0.8, 0.156);
  hipComplex a(jx, jy);
  int i = 0;
  for (i = 0; i < 200; i++)
  {
    a = a * a + c;
    if (a.magnitude2() > 1000)
    {
      return 0;
    }
  }
  return 1;
}

__global__ void kernel(unsigned char *ptr)
{
  // map from threadIdx/BlockIdx to pixel position
  int x = blockIdx.x;
  int y = blockIdx.y;
  int offset = x + y * gridDim.x;

  // now calculate the value at that position
  int juliaValue = julia(x, y);

  // We have 4 different channels
  ptr[offset * 4 + 0] = 0;
  ptr[offset * 4 + 1] = 255 * juliaValue;
  ptr[offset * 4 + 2] = 0;
  ptr[offset * 4 + 3] = 200;
}

int main(void)
{
  CPUBitmap bitmap(DIM, DIM);
  unsigned char *dev_bitmap;
  HANDLE_ERROR(hipMalloc((void **)&dev_bitmap, bitmap.image_size()));

  dim3 grid(DIM, DIM);
  kernel<<<grid, 1>>>(dev_bitmap);
  HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));
  bitmap.display_and_exit();
  HANDLE_ERROR(hipFree(dev_bitmap));
}